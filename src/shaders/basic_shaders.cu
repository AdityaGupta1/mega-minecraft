#include "hip/hip_runtime.h"
#include "shader_commons.h"

/*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
extern "C" __constant__ OptixParams params;


static __forceinline__ __device__
void* unpackPointer(uint32_t i0, uint32_t i1)
{
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__
void  packPointer(void* ptr, uint32_t& i0, uint32_t& i1)
{
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T* getPRD()
{
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

extern "C" __global__ void __raygen__render() {
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;

    const auto& camera = params.camera;
    float3 pixelColorPRD = make_float3(0.f);

    uint32_t u0, u1;
    packPointer(&pixelColorPRD, u0, u1);

    float3 rayDir = normalize(camera.forward
        - camera.right * camera.pixelLength.x * ((float)ix - (float)params.windowSize.x * 0.5f)
        - camera.up * camera.pixelLength.y * -((float)iy - (float)params.windowSize.y * 0.5f)
    );

    optixTrace(params.rootHandle,
        camera.position,
        rayDir,
        0.f,    // tmin
        1e20f,  // tmax
        0.0f,   // rayTime
        OptixVisibilityMask(255),
        OPTIX_RAY_FLAG_NONE,  //OPTIX_RAY_FLAG_NONE,
        0,  // SBT offset
        1,  // SBT stride
        0,  // missSBTIndex
        u0, u1);

    const int r = int(255.99f * pixelColorPRD.x);
    const int g = int(255.99f * pixelColorPRD.y);
    const int b = int(255.99f * pixelColorPRD.z);

    //const int r = int(255.99f * (ix / (float)(params.windowSize.x)));
    //const int g = int(255.99f * (iy / (float)(params.windowSize.y)));
    //const int b = 0;

    // and write to frame buffer ...
    const uint32_t fbIndex = ix + iy * params.windowSize.x;

    const uint32_t rgba = 0xff000000
        | (r << 0) | (g << 8) | (b << 16);    

    params.frame.colorBuffer[fbIndex] = rgba;
}

extern "C" __global__ void __miss__radiance() {
    float3& prd = *(float3*)getPRD<float3>();
    prd = make_float3(0.5f, 0.8f, 1.0f) * 0.4f;
}

extern "C" __global__ void __closesthit__radiance() {
    const ChunkData& chunkData = *(const ChunkData*)optixGetSbtDataPointer();

    const int primID = optixGetPrimitiveIndex();

    const uint3 vIdx = chunkData.idx[primID];

    const Vertex& v1 = chunkData.verts[vIdx.x];
    const Vertex& v2 = chunkData.verts[vIdx.y];
    const Vertex& v3 = chunkData.verts[vIdx.z];

    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    float2 uv = (1.f - u - v) * v1.uv + u * v2.uv + v * v3.uv;
    float4 diffuseCol = tex2D<float4>(chunkData.tex_diffuse, uv.x, uv.y);

    float3& prd = *(float3*)getPRD<float3>();
    prd = make_float3(diffuseCol);
}

extern "C" __global__ void __anyhit__radiance()
{

}

extern "C" __global__ void __exception__all()
{
    // This assumes that the launch dimensions are matching the size of the output buffer.

    const uint3 theLaunchIndex = optixGetLaunchIndex();

    const int theExceptionCode = optixGetExceptionCode();
    printf("Exception %d at (%u, %u)\n", theExceptionCode, theLaunchIndex.x, theLaunchIndex.y);
}