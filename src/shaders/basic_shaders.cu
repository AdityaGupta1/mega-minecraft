#include "hip/hip_runtime.h"
#include <optix.h>
#include "shader_commons.h"
#include "random_number_generators.h"

#define PI                3.14159265358979323846264338327f
#define TWO_PI            6.28318530717958647692528676655f
#define PI_OVER_TWO       1.57079632679489661923132169163f
#define PI_OVER_FOUR      0.78539816339744830961566084581f
#define SQRT_2            1.41421356237309504880168872420f
#define SQRT_ONE_THIRD    0.57735026918962576450914878050f

#define NUM_SAMPLES 2
#define MAX_RAY_DEPTH 5

/*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
extern "C" __constant__ OptixParams params;

static __forceinline__ __device__
void* unpackPointer(uint32_t i0, uint32_t i1)
{
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__
void  packPointer(void* ptr, uint32_t& i0, uint32_t& i1)
{
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T* getPRD()
{
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

extern "C" __global__ void __raygen__render() {
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;

    const int dx = optixGetLaunchDimensions().x;

    PRD prd;
    prd.seed = tea<4>(iy * dx + ix, params.frame.frameId);

    const auto& camera = params.camera;
    float2 squareSample = rng2(prd.seed);
    float3 rayDir = normalize(camera.forward
        - camera.right * camera.pixelLength.x * ((float)ix - (float)params.windowSize.x * 0.5f + squareSample.x)
        - camera.up * camera.pixelLength.y * -((float)iy - (float)params.windowSize.y * 0.5f + squareSample.y)
    );

    uint32_t u0, u1;
    packPointer(&prd, u0, u1);

    float3 finalColor = make_float3(0);
    float3 finalAlbedo = make_float3(0);
    float3 finalNormal = make_float3(0);

    for (int sample = 0; sample < NUM_SAMPLES; ++sample)
    {
        prd.isDone = false;
        prd.needsFirstHitData = true;

        prd.isect.pos = camera.position;
        prd.isect.newDir = rayDir;

        prd.pixelColor = make_float3(1.f, 1.f, 1.f);

        for (int depth = 0; depth < MAX_RAY_DEPTH && !prd.isDone; ++depth)
        {
            optixTrace(params.rootHandle,
                prd.isect.pos,
                prd.isect.newDir,
                0.f,    // tmin
                1e20f,  // tmax
                0.0f,   // rayTime
                OptixVisibilityMask(255),
                OPTIX_RAY_FLAG_NONE,  // OPTIX_RAY_FLAG_NONE,
                0,  // SBT offset
                1,  // SBT stride
                0,  // missSBTIndex
                u0, u1);
        }

        if (!prd.isDone) // reached max depth and didn't hit a light
                         // TODO: sample direct lighting at this point
        {
            prd.pixelColor = make_float3(0.f);
        }

        finalColor += prd.pixelColor;
        finalAlbedo += prd.pixelAlbedo;
        finalNormal += prd.pixelNormal;
    }

    finalColor /= NUM_SAMPLES;
    finalAlbedo /= NUM_SAMPLES;
    finalNormal /= NUM_SAMPLES;

    // accumulate colors
    const uint32_t fbIndex = ix + iy * params.windowSize.x;

    int frameId = params.frame.frameId;
    if (frameId > 0) {
        float multiplier = 1.f / (frameId + 1.f);
        finalColor = (finalColor + frameId * make_float3(params.frame.colorBuffer[fbIndex])) * multiplier;
        finalAlbedo = (finalAlbedo + frameId * make_float3(params.frame.albedoBuffer[fbIndex])) * multiplier;
        finalNormal = (finalNormal + frameId * make_float3(params.frame.normalBuffer[fbIndex])) * multiplier;
    }

    params.frame.colorBuffer[fbIndex] = make_float4(finalColor, 1.f);
    params.frame.albedoBuffer[fbIndex] = make_float4(finalAlbedo, 1.f);
    params.frame.normalBuffer[fbIndex] = make_float4(finalNormal, 1.f);
}

static __forceinline__ __device__
const ChunkData& getChunkData()
{
    return *(const ChunkData*)optixGetSbtDataPointer();
}

static __forceinline__ __device__
void getVerts(const ChunkData& chunkData, Vertex* v1, Vertex* v2, Vertex* v3)
{
    const int primID = optixGetPrimitiveIndex();
    const uint3 vIdx = chunkData.idx[primID];
    *v1 = chunkData.verts[vIdx.x];
    *v2 = chunkData.verts[vIdx.y];
    *v3 = chunkData.verts[vIdx.z];
}

static __forceinline__ __device__
float3 getBarycentricCoords()
{
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;
    return make_float3(1.f - u - v, u, v);
}

extern "C" __global__ void __miss__radiance()
{
    const float3 rayDir = optixGetWorldRayDirection();

    float3 skyColor;
    if (dot(rayDir, params.sunDir) > 0.99f)
    {
        skyColor = make_float3(1.0f, 0.8f, 0.6f) * 1.1f;
    }
    else
    {
        skyColor = make_float3(0.5f, 0.8f, 1.0f) * 0.3f;
    }

    PRD& prd = *getPRD<PRD>();
    prd.isDone = true;
    prd.pixelColor *= skyColor;
    if (prd.needsFirstHitData)
    {
        prd.needsFirstHitData = false;
        prd.pixelAlbedo = skyColor;
        prd.pixelNormal = -rayDir;
    }
}

__device__ float3 calculateDirectionNotNormal(const float3 normal)
{
    if (fabs(normal.x) < SQRT_ONE_THIRD)
    {
        return make_float3(1, 0, 0);
    }
    else if (fabs(normal.y) < SQRT_ONE_THIRD)
    {
        return make_float3(0, 1, 0);
    }
    else
    {
        return make_float3(0, 0, 1);
    }
}

__device__ float3 calculateRandomDirectionInHemisphere(float3 normal, float2 sample)
{
    const float up = sqrt(sample.x); // cos(theta)
    const float over = sqrt(1.f - sample.x); // sin(theta)
    const float around = sample.y * TWO_PI;

    // Use not-normal direction to generate two perpendicular directions
    const float3 perpendicularDirection1 = normalize(cross(normal, calculateDirectionNotNormal(normal)));
    const float3 perpendicularDirection2 = normalize(cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

extern "C" __global__ void __closesthit__radiance() {
    const ChunkData& chunkData = getChunkData();
    Vertex v1, v2, v3;
    getVerts(chunkData, &v1, &v2, &v3);

    const float3 bary = getBarycentricCoords();
    float2 uv = bary.x * v1.uv + bary.y * v2.uv + bary.z * v3.uv;
    float3 diffuseCol = make_float3(tex2D<float4>(chunkData.tex_diffuse, uv.x, uv.y));

    const float3 rayDir = optixGetWorldRayDirection();
    float3 isectPos = optixGetWorldRayOrigin() + rayDir * optixGetRayTmax();

    PRD& prd = *getPRD<PRD>();

    float3 nor = normalize(bary.x * v1.nor + bary.y * v2.nor + bary.z * v3.nor);
    float3 newDir = calculateRandomDirectionInHemisphere(nor, rng2(prd.seed));
    // don't multiply by lambert term since it's canceled out by PDF for uniform hemisphere sampling

    prd.isect.pos = isectPos + nor * 0.001f;
    prd.isect.newDir = newDir;

    prd.pixelColor *= diffuseCol;
    if (prd.needsFirstHitData)
    {
        prd.needsFirstHitData = false;
        prd.pixelAlbedo = diffuseCol;
        prd.pixelNormal = nor;
    }
}

extern "C" __global__ void __anyhit__radiance()
{
    const ChunkData& chunkData = getChunkData();
    Vertex v1, v2, v3;
    getVerts(chunkData, &v1, &v2, &v3);

    const float3 bary = getBarycentricCoords();
    float2 uv = bary.x * v1.uv + bary.y * v2.uv + bary.z * v3.uv;
    float4 diffuseCol = tex2D<float4>(chunkData.tex_diffuse, uv.x, uv.y);

    if (diffuseCol.w == 0.f)
    {
        optixIgnoreIntersection();
    }
    // TODO: figure out whether to use normal faceforwards before re-enabling this
    //else
    //{
    //    PRD& prd = *getPRD<PRD>();
    //    if (rng(prd.seed) >= diffuseCol.w)
    //    {
    //        optixIgnoreIntersection();
    //    }
    //}
}

extern "C" __global__ void __exception__all()
{
    // This assumes that the launch dimensions are matching the size of the output buffer.

    const uint3 theLaunchIndex = optixGetLaunchIndex();

    const int theExceptionCode = optixGetExceptionCode();
    printf("Exception %d at (%u, %u)\n", theExceptionCode, theLaunchIndex.x, theLaunchIndex.y);
}