#include "hip/hip_runtime.h"
#include <optix.h>
#include "shader_commons.h"
#include "random_number_generators.h"

#define PI                3.14159265358979323846264338327f
#define TWO_PI            6.28318530717958647692528676655f
#define PI_OVER_TWO       1.57079632679489661923132169163f
#define PI_OVER_FOUR      0.78539816339744830961566084581f
#define SQRT_2            1.41421356237309504880168872420f
#define SQRT_ONE_THIRD    0.57735026918962576450914878050f
#define INV_PI            0.31830988618379067153776752674f

#define NUM_SAMPLES 4
#define MAX_RAY_DEPTH 2

/*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
extern "C" __constant__ OptixParams params;

static __forceinline__ __device__
void* unpackPointer(uint32_t i0, uint32_t i1)
{
    const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
    void* ptr = reinterpret_cast<void*>(uptr);
    return ptr;
}

static __forceinline__ __device__
void  packPointer(void* ptr, uint32_t& i0, uint32_t& i1)
{
    const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T* getPRD()
{
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

static __forceinline__ __device__ 
float powerHeuristics(int nf, float pdf_f, int ng, float pdf_g) {
    float f = nf * pdf_f;
    float g = ng * pdf_g;
    return f * f / (f * f + g * g);
}

__device__ float3 calculateDirectionNotNormal(const float3 normal)
{
    if (fabs(normal.x) < SQRT_ONE_THIRD)
    {
        return make_float3(1, 0, 0);
    }
    else if (fabs(normal.y) < SQRT_ONE_THIRD)
    {
        return make_float3(0, 1, 0);
    }
    else
    {
        return make_float3(0, 0, 1);
    }
}

__device__ float3 calculateRandomDirectionInHemisphere(float3 normal, float2 sample)
{
    const float up = sqrt(sample.x); // cos(theta)
    const float over = sqrt(1.f - sample.x); // sin(theta)
    const float around = sample.y * TWO_PI;

    // Use not-normal direction to generate two perpendicular directions
    const float3 perpendicularDirection1 = normalize(cross(normal, calculateDirectionNotNormal(normal)));
    const float3 perpendicularDirection2 = normalize(cross(normal, perpendicularDirection1));

    return up * normal
        + cos(around) * over * perpendicularDirection1
        + sin(around) * over * perpendicularDirection2;
}

__device__ float3 sampleSun(float2 sample)
{
    // find radius and theta in sun space

    // Use not-normal direction to generate two perpendicular directions
    const float3 normal = params.sunDir;

    const float3 perpendicularDirection1 = normalize(cross(normal, calculateDirectionNotNormal(normal)));
    const float3 perpendicularDirection2 = normalize(cross(normal, perpendicularDirection1));

    const float around = sample.y * TWO_PI; // theta

    float3 dir = cos(around) * perpendicularDirection1 + sin(around) * perpendicularDirection2;

    float3 normal2 = normal * normal;
    float max_r = (0.99f - normal2.x - normal2.y - normal2.z) / dot(dir, normal);

    // TODO: so I cheesed this a bit for now but 0.01 should be max_r

    return normal + sample.x * 0.01 * dir;
}

extern "C" __global__ void __raygen__render() {
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;

    const int dx = optixGetLaunchDimensions().x;

    PRD prd;
    prd.seed = tea<4>(iy * dx + ix, params.frame.frameId);

    const auto& camera = params.camera;
    float2 squareSample = rng2(prd.seed);
    float3 rayDir = normalize(camera.forward
        - camera.right * camera.pixelLength.x * ((float)ix - (float)params.windowSize.x * 0.5f + squareSample.x)
        - camera.up * camera.pixelLength.y * -((float)iy - (float)params.windowSize.y * 0.5f + squareSample.y)
    );

    uint32_t u0, u1;
    packPointer(&prd, u0, u1);

    float3 finalColor = make_float3(0);
    float3 finalAlbedo = make_float3(0);
    float3 finalNormal = make_float3(0);

    for (int sample = 0; sample < NUM_SAMPLES; ++sample)
    {
        prd.isDone = false;
        prd.foundLightSource = false;
        prd.rayColor = make_float3(1.f);
        prd.pixelColor = make_float3(0.f);
        prd.isect.pos = camera.position;
        prd.isect.newDir = rayDir;

        for (int depth = 0; depth < MAX_RAY_DEPTH && !prd.isDone; ++depth)
        {
            // 1. BSDF

            optixTrace(params.rootHandle,
                prd.isect.pos,
                prd.isect.newDir,
                0.f,    // tmin
                1e20f,  // tmax
                0.0f,   // rayTime
                OptixVisibilityMask(255),
                OPTIX_RAY_FLAG_NONE,  // OPTIX_RAY_FLAG_NONE,
                0,  // SBT offset
                1,  // SBT stride
                0,  // missSBTIndex
                u0, u1);

            if (!prd.isDone) {
                // MIS: sample light source
                // 2. pdf from Sun & random point on sun
                float2 xi = rng2(prd.seed);

                float3 random_d = sampleSun(xi);

                // 3. test sun intersection
                prd.foundLightSource = true;

                optixTrace(params.rootHandle,
                    prd.isect.pos,
                    random_d,
                    0.f,    // tmin
                    1e20f,  // tmax
                    0.0f,   // rayTime
                    OptixVisibilityMask(255),
                    OPTIX_RAY_FLAG_NONE,  // OPTIX_RAY_FLAG_NONE,
                    1,  // SBT offset
                    1,  // SBT stride
                    0,  // missSBTIndex
                    u0, u1);

                // TODO: later, find pdf for each material, using default for now
                // heuristics uses next direction & sun direction pdfs

                if (prd.foundLightSource) {
                    float pdf_material = INV_PI * dot(random_d, prd.isect.newDir);
                    float3 col = powerHeuristics(1, 1.f, 1, pdf_material) * prd.rayColor;
                    prd.foundLightSource = false;
                    prd.pixelColor += col * prd.rayColor;
                }
                
            }
            
        }

        if (!prd.isDone) // reached max depth and didn't hit a light
                         // TODO: sample direct lighting at this point
        {
            // Direct Lighting
            prd.isect.newDir = sampleSun(rng2(prd.seed));
            optixTrace(params.rootHandle,
                prd.isect.pos,
                prd.isect.newDir,
                0.f,    // tmin
                1e20f,  // tmax
                0.0f,   // rayTime
                OptixVisibilityMask(255),
                OPTIX_RAY_FLAG_NONE,  // OPTIX_RAY_FLAG_NONE,
                0,  // SBT offset
                1,  // SBT stride
                0,  // missSBTIndex
                u0, u1);
            if (!prd.foundLightSource) {
                prd.pixelColor = make_float3(0.f);
            }
        }

        finalColor += prd.pixelColor;
        finalAlbedo += prd.pixelAlbedo;
        finalNormal += prd.pixelNormal;
    }

    finalColor /= NUM_SAMPLES;
    finalAlbedo /= NUM_SAMPLES;
    finalNormal /= NUM_SAMPLES;

    // accumulate colors
    const uint32_t fbIndex = ix + iy * params.windowSize.x;

    float4 cumColor = make_float4(finalColor, 1.f);

    int frameId = params.frame.frameId;
    if (frameId > 0) {
        cumColor += float(frameId) * params.frame.colorBuffer[fbIndex];
        cumColor /= (frameId + 1.f);
    }

    params.frame.colorBuffer[fbIndex] = cumColor;
}

static __forceinline__ __device__
const ChunkData& getChunkData()
{
    return *(const ChunkData*)optixGetSbtDataPointer();
}

static __forceinline__ __device__
void getVerts(const ChunkData& chunkData, Vertex* v1, Vertex* v2, Vertex* v3)
{
    const int primID = optixGetPrimitiveIndex();
    const uint3 vIdx = chunkData.idx[primID];
    *v1 = chunkData.verts[vIdx.x];
    *v2 = chunkData.verts[vIdx.y];
    *v3 = chunkData.verts[vIdx.z];
}

static __forceinline__ __device__
float3 getBarycentricCoords()
{
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;
    return make_float3(1.f - u - v, u, v);
}

extern "C" __global__ void __miss__radiance()
{
    const float3 rayDir = optixGetWorldRayDirection();
    PRD& prd = *getPRD<PRD>();

    float3 skyColor;
    float d = dot(rayDir, params.sunDir);
    if (d > 0.99f)
    {
        float hue = dot(params.sunDir, make_float3(0.f, 1.f, 0.f));
        skyColor = make_float3(1.0f, 0.6f + 0.2f * hue, 0.4f + 0.2f * hue) * (1.f - 5000.f * (1.f - d) * (1.f - d));
        prd.foundLightSource = true;
    }
    else
    {
        skyColor = make_float3(0.5f, 0.8f, 1.0f) * 0.2f;
    }

    prd.pixelColor += skyColor * prd.rayColor;
    prd.isDone = true;
}

extern "C" __global__ void __closesthit__radiance() {
    PRD& prd = *getPRD<PRD>();

    const ChunkData& chunkData = getChunkData();
    Vertex v1, v2, v3;
    getVerts(chunkData, &v1, &v2, &v3);

    const float3 bary = getBarycentricCoords();
    float2 uv = bary.x * v1.uv + bary.y * v2.uv + bary.z * v3.uv;
    float4 diffuseCol = tex2D<float4>(chunkData.tex_diffuse, uv.x, uv.y);

    const float3 rayDir = optixGetWorldRayDirection();
    float3 isectPos = optixGetWorldRayOrigin() + rayDir * optixGetRayTmax();

    float3 nor = normalize(bary.x * v1.nor + bary.y * v2.nor + bary.z * v3.nor);
    float3 newDir = calculateRandomDirectionInHemisphere(nor, rng2(prd.seed));

    // don't multiply by lambert term since it's canceled out by PDF for uniform hemisphere sampling

    // prd.pixelColor *= prd.rayColor;
    prd.rayColor *= make_float3(diffuseCol);
    prd.isect.pos = isectPos + nor * 0.001f;
    prd.isect.newDir = newDir;
}

extern "C" __global__ void __anyhit__radiance()
{
    const ChunkData& chunkData = getChunkData();
    Vertex v1, v2, v3;
    getVerts(chunkData, &v1, &v2, &v3);

    const float3 bary = getBarycentricCoords();
    float2 uv = bary.x * v1.uv + bary.y * v2.uv + bary.z * v3.uv;
    float4 diffuseCol = tex2D<float4>(chunkData.tex_diffuse, uv.x, uv.y);

    if (diffuseCol.w == 0.f)
    {
        optixIgnoreIntersection();
    }
    // TODO: figure out whether to use normal faceforwards before re-enabling this
    //else
    //{
    //    PRD& prd = *getPRD<PRD>();
    //    if (rng(prd.seed) >= diffuseCol.w)
    //    {
    //        optixIgnoreIntersection();
    //    }
    //}
}

extern "C" __global__ void __anyhit__shadow() {
    PRD& prd = *getPRD<PRD>();
    prd.foundLightSource = false;
    optixTerminateRay();
}

extern "C" __global__ void __exception__all()
{
    // This assumes that the launch dimensions are matching the size of the output buffer.

    const uint3 theLaunchIndex = optixGetLaunchIndex();

    const int theExceptionCode = optixGetExceptionCode();
    printf("Exception %d at (%u, %u)\n", theExceptionCode, theLaunchIndex.x, theLaunchIndex.y);
}