#include "hip/hip_runtime.h"
#include "chunk.hpp"

#include "rendering/structs.hpp"
#include "rendering/renderingUtils.hpp"
#include "util/enums.hpp"
#include "cuda/cuda_utils.hpp"
#include <glm/gtc/noise.hpp>

#include <iostream>

Chunk::Chunk(ivec2 worldChunkPos)
    : worldChunkPos(worldChunkPos)
{
    std::fill_n(heightfield.begin(), 256, 0);
    std::fill_n(blocks.begin(), 65536, Block::AIR);
}

ChunkState Chunk::getState()
{
    return this->state;
}

void Chunk::setState(ChunkState newState)
{
    this->state = newState;
    this->readyForQueue = true;
}

bool Chunk::isReadyForQueue()
{
    return this->readyForQueue;
}

void Chunk::setNotReadyForQueue()
{
    this->readyForQueue = false;
}

__host__ __device__
int posToIndex(const int x, const int y, const int z)
{
    return x + 16 * z + 256 * y;
}

__host__ __device__
int posToIndex(const ivec3 pos)
{
    return posToIndex(pos.x, pos.y, pos.z);
}

__host__ __device__
int posToIndex(const int x, const int z)
{
    return x + 16 * z;
}

__host__ __device__
int posToIndex(const ivec2 pos)
{
    return posToIndex(pos.x, pos.y);
}

void Chunk::dummyFill()
{
    for (int z = 0; z < 16; ++z)
    {
        for (int x = 0; x < 16; ++x)
        {
            int height = 48 + (x + z) / 2;
            if ((x + z) % 4 == 1)
            {
                height += 10;
            }
            for (int y = 0; y < height; ++y)
            {
                Block block = Block::STONE;
                if (y == height - 1)
                {
                    block = Block::GRASS;
                }
                else if (y < height - 1 && y >= height - 4)
                {
                    block = Block::DIRT;
                }

                this->blocks[posToIndex(x, y, z)] = block;
            }
        }
    }
}

__device__
float dummyNoise(vec2 pos)
{
    pos *= 0.02f;

    float fbm = 0.f;
    float amplitude = 1.f;
    for (int i = 0; i < 4; ++i)
    {
        amplitude *= 0.5f;
        pos *= 2.f;
        fbm += amplitude * glm::simplex(pos);
    }

    return 64.f + 6.f * fbm;
}

__global__ void kernDummyGenerateHeightfield(unsigned char* heightfield, ivec2 worldBlockPos)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int z = (blockIdx.y * blockDim.y) + threadIdx.y;

    const int idx = posToIndex(x, z);

    const vec2 worldPos = vec2(worldBlockPos.x + x, worldBlockPos.y + z);

    int height = (int)dummyNoise(worldPos);

    heightfield[idx] = height;
}

__global__ void kernDummyFill(Block* blocks, unsigned char* heightfield)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int z = (blockIdx.z * blockDim.z) + threadIdx.z;

    const int idx = posToIndex(x, y, z);

    const unsigned char height = heightfield[posToIndex(x, z)]; // TODO: when implementing for real, use shared memory to load heightfield

    Block block = Block::STONE;
    if (y > height)
    {
        block = Block::AIR;
    }
    else if (y == height)
    {
        block = Block::SAND;
    }
    else if (y >= height - 3)
    {
        block = Block::DIRT;
    }

    blocks[idx] = block;
}

void Chunk::dummyFillCUDA(Block* dev_blocks, unsigned char* dev_heightfield)
{
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(2, 2);

    const dim3 blockSize3d(1, 256, 1);
    const dim3 blocksPerGrid3d(16, 1, 16);

    hipEvent_t start, mid, stop;
    hipEventCreate(&start);
    hipEventCreate(&mid);
    hipEventCreate(&stop);

    hipEventRecord(start);
    kernDummyGenerateHeightfield<<<blocksPerGrid2d, blockSize2d>>>(dev_heightfield, this->worldChunkPos * 16);
    CudaUtils::checkCUDAError("kern generate heightfield failed");
    hipEventRecord(mid);

    hipDeviceSynchronize();

    // TODO: when implementing for real, the two kernels will happen separately; will probably need to copy heightfield back to GPU before running this kernel
    kernDummyFill<<<blocksPerGrid3d, blockSize3d>>>(dev_blocks, dev_heightfield);
    CudaUtils::checkCUDAError("kern fill failed");
    
    hipMemcpy(this->blocks.data(), dev_blocks, 65536 * sizeof(Block), hipMemcpyDeviceToHost);
    CudaUtils::checkCUDAError("hipMemcpy failed");
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    //float milliseconds = 0;
    //hipEventElapsedTime(&milliseconds, start, stop);
    //std::cout << "full ms elapsed: " << milliseconds << std::endl;
    //hipEventElapsedTime(&milliseconds, start, mid);
    //std::cout << "mid ms elapsed: " << milliseconds << std::endl;
    //std::cout << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(mid);
    hipEventDestroy(stop);
}

static const std::array<vec3, 24> directionVertPositions = {
    vec3(0, 0, 1), vec3(1, 0, 1), vec3(1, 1, 1), vec3(0, 1, 1),
    vec3(1, 0, 1), vec3(1, 0, 0), vec3(1, 1, 0), vec3(1, 1, 1),
    vec3(1, 0, 0), vec3(0, 0, 0), vec3(0, 1, 0), vec3(1, 1, 0),
    vec3(0, 0, 0), vec3(0, 0, 1), vec3(0, 1, 1), vec3(0, 1, 0),
    vec3(0, 1, 1), vec3(1, 1, 1), vec3(1, 1, 0), vec3(0, 1, 0),
    vec3(0, 0, 0), vec3(1, 0, 0), vec3(1, 0, 1), vec3(0, 0, 1)
};

static const std::array<vec2, 16> uvOffsets = {
    vec2(0, 0), vec2(0.0625f, 0), vec2(0.0625f, 0.0625f), vec2(0, 0.0625f)
};

float randFromPosDir(ivec3 blockPos, int dir)
{
    return fract(sin(dot(vec4(vec3(blockPos), dir), vec4(453.29f, 817.46f, 296.14f, 572.85f))));
}

float randFromRand(float rand)
{
    return fract(sin(rand * 134.78f));
}

void Chunk::createVBOs()
{
    idx.clear();
    verts.clear();

    idxCount = 0;

    for (int y = 0; y < 256; ++y)
    {
        for (int z = 0; z < 16; ++z)
        {
            for (int x = 0; x < 16; ++x)
            {
                ivec3 thisPos = ivec3(x, y, z);
                Block thisBlock = blocks[posToIndex(thisPos)];

                if (thisBlock == Block::AIR)
                {
                    continue;
                }

                BlockData thisBlockData = BlockUtils::getBlockData(thisBlock);

                for (int dirIdx = 0; dirIdx < 6; ++dirIdx)
                {
                    const auto& direction = DirectionEnums::dirVecs[dirIdx];
                    ivec3 neighborPos = thisPos + direction;
                    Chunk* neighborPosChunk = this;
                    Block neighborBlock;

                    if (neighborPos.y >= 0 && neighborPos.y < 256)
                    {
                        if (neighborPos.x < 0)
                        {
                            neighborPosChunk = neighbors[3];
                            neighborPos.x += 16;
                        }
                        else if (neighborPos.x >= 16)
                        {
                            neighborPosChunk = neighbors[1];
                            neighborPos.x -= 16;
                        }
                        else if (neighborPos.z < 0)
                        {
                            neighborPosChunk = neighbors[2];
                            neighborPos.z += 16;
                        }
                        else if (neighborPos.z >= 16)
                        {
                            neighborPosChunk = neighbors[0];
                            neighborPos.z -= 16;
                        }

                        if (neighborPosChunk == nullptr)
                        {
                            continue;
                        }

                        neighborBlock = neighborPosChunk->blocks[posToIndex(neighborPos)];

                        if (neighborBlock != Block::AIR) // TODO: this will get more complicated with transparent and non-cube blocks
                        {
                            continue;
                        }
                    }

                    int idx1 = verts.size();

                    const auto& thisUvs = thisBlockData.uvs;
                    vec2 uvStart;
                    bool uvRandRot;
                    bool uvRandFlip;
                    switch (direction.y)
                    {
                    case 1:
                        uvStart = thisUvs.top;
                        uvRandRot = thisUvs.randRotTop;
                        uvRandFlip = thisUvs.randFlipTop;
                        break;
                    case -1:
                        uvStart = thisUvs.bottom;
                        uvRandRot = thisUvs.randRotBottom;
                        uvRandFlip = thisUvs.randFlipBottom;
                        break;
                    case 0:
                        uvStart = thisUvs.side;
                        uvRandRot = thisUvs.randRotSide;
                        uvRandFlip = thisUvs.randFlipSide;
                        break;
                    }

                    int uvStartIdx = 0;
                    int uvFlipIdx = -1;
                    if (uvRandRot || uvRandFlip)
                    {
                        float rand = randFromPosDir(thisPos, dirIdx);
                        if (uvRandRot)
                        {
                            uvStartIdx = (int)(rand * 4.f);
                            rand = randFromRand(rand);
                        }
                        if (uvRandFlip)
                        {
                            uvFlipIdx = (int)(rand * 4.f);
                        }
                    }

                    for (int j = 0; j < 4; ++j)
                    {
                        verts.emplace_back();
                        Vertex& vert = verts.back();

                        vert.pos = vec3(thisPos) + directionVertPositions[dirIdx * 4 + j];

                        vec2 uvOffset = uvOffsets[(uvStartIdx + j) % 4];
                        if (uvFlipIdx != -1)
                        {
                            if (uvFlipIdx & 1)
                            {
                                uvOffset.x = 0.0625f - uvOffset.x;
                            }
                            if (uvFlipIdx & 2)
                            {
                                uvOffset.y = 0.0625f - uvOffset.y;
                            }
                        }
                        vert.uv = uvStart + uvOffset;
                    }

                    idx.push_back(idx1);
                    idx.push_back(idx1 + 1);
                    idx.push_back(idx1 + 2);
                    idx.push_back(idx1);
                    idx.push_back(idx1 + 2);
                    idx.push_back(idx1 + 3);
                }
            }
        }
    }
}

void Chunk::bufferVBOs()
{
    idxCount = idx.size();

    generateIdx();
    bindIdx();
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, idx.size() * sizeof(GLuint), idx.data(), GL_STATIC_DRAW);

    generateVerts();
    bindVerts();
    glBufferData(GL_ARRAY_BUFFER, verts.size() * sizeof(Vertex), verts.data(), GL_STATIC_DRAW);

    idx.clear();
    verts.clear();
}